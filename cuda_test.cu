#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>
#include <string>
#include <hip/hip_runtime.h>
#include "cuda_test.h"


// Wrappers around cuda calls to stop upon runtime errors
inline void __cuda_check(hipError_t code, const char* fn, const char* file, int line) {
    if (code != hipSuccess) {
        std::printf("\nCUDA error %d running %s\n%s\nIn %s line %d\n\n", code, fn, hipGetErrorString(code), file, line);
        std::exit(1);
    }
}
#define CUDA_CHECK(op) __cuda_check((op), #op, __FILE__, __LINE__)
#define CUDA_CHECK_K __cuda_check(hipPeekAtLastError(), "kernel", __FILE__, __LINE__)



__global__ void vadd_k(float* a, float* b, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        a[idx] += b[idx];
    }
}

void testCuda (int n, int blockSize) {
    std::printf("Starting CUDA test, N = %d, block size = %d.\n", n, blockSize);
    
    // Alloc resources
    float* x = (float*)std::malloc(n * sizeof(float));
    float* y = (float*)std::malloc(n * sizeof(float));
    float* x_d;
    float* y_d;
    CUDA_CHECK(hipMalloc(&x_d, n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&y_d, n*sizeof(float)));
    for (int i = 0; i < n; ++i){
        x[i] = 3;
        y[i] = 7;
    }
    
    // Copy to device
    CUDA_CHECK(hipMemcpy(x_d, x, n*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(y_d, y, n*sizeof(float), hipMemcpyHostToDevice));
    
    // Run kernel
    vadd_k<<<n/blockSize+(n%blockSize!=0), blockSize>>>(x_d, y_d, n);
    CUDA_CHECK_K;
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy results back to host
    CUDA_CHECK(hipMemcpy(x, x_d, n*sizeof(float), hipMemcpyDeviceToHost));
    
    // Check results
    std::printf("Expecting 10s:\n");
    for (int i = 0; i < n && i < 100; ++i) {
        std::printf("%f ", x[i]);
    }
    std::printf("\n\n");
    bool valid = true;
    for (int i = 0; i < n; ++i) {
        if (x[i] != 10) {
            valid = false;
            break;
        }
    }
    if (valid) {
        std::printf("Valid across all %d elements.\n", n);
    } else {
        std::printf("Test was invalid on at least one element.\n");
    }
    
    // Clean up
    CUDA_CHECK(hipFree(x_d));
    CUDA_CHECK(hipFree(y_d));
    std::free(x);
    std::free(y);
    std::printf("Finished CUDA test.\n");
}
