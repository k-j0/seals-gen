
#include "cuda_utils.h"
#include "cuda_info.h"

std::string getCudaInfo () {
    
    std::string out = "CUDA enabled, ";
    
    int numDevices;
    CUDA_CHECK(hipGetDeviceCount(&numDevices));
    out += std::to_string(numDevices) + " device" + (numDevices != 1 ? "s" : "") + ":";
    
    for (int i = 0; i < numDevices; ++i) {
        hipDeviceProp_t props;
        CUDA_CHECK(hipGetDeviceProperties(&props, i));
        out += "\n\t- " + std::string(props.name);
        out += ", SM " + std::to_string(props.major) + "." + std::to_string(props.minor);
        if (props.isMultiGpuBoard) {
            out += " (Multi-GPU board)";
        }
        if (props.integrated) {
            out += " (integrated)";
        }
        out += "\n\t\t" + std::to_string(props.clockRate / 1000) + "MHz (memory: " + std::to_string(props.memoryClockRate / 1000) + "MHz)";
        out += "\n\t\t" + std::to_string(props.asyncEngineCount) + " async engines, " + std::to_string(props.multiProcessorCount) + " multiprocessors";
    }
    
    return out;
}
